#include "hip/hip_runtime.h"
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#include <hip/hip_fp16.h>
__device__ half max(half a, half b)
{
  return __hgt(__half(a), __half(b)) ? a : b;
}
__device__ half min(half a, half b)
{
  return __hlt(__half(a), __half(b)) ? a : b;
}
#else

typedef unsigned short uint16_t;
typedef unsigned char uint8_t;
typedef signed char int8_t;
typedef int int32_t;
typedef unsigned long long uint64_t;
typedef unsigned int uint32_t;

#define TVM_FORCE_INLINE inline __attribute__((always_inline))
#define TVM_XINLINE TVM_FORCE_INLINE __device__ __host__
#define TVM_ALIGNED(x) __attribute__ ((aligned(x)))
#define TVM_HALF_OPERATOR(RTYPE, OP)                              \
  TVM_XINLINE RTYPE operator OP (half a, half b) {                \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (half a, T b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (T a, half b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }

#define TVM_HALF_ASSIGNOP(AOP, OP)                                \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const T& a) {                    \
    return *this = half(float(*this) OP float(a));                \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const volatile T& a) volatile {  \
    return *this = half(float(*this) OP float(a));                \
  }

class TVM_ALIGNED(2) half {
 public:
  uint16_t half_;

  static TVM_XINLINE half Binary(uint16_t value) {
    half res;
    res.half_ = value;
    return res;
  }

  TVM_XINLINE half() {}

  TVM_XINLINE half(const float& value) { constructor(value); }
  TVM_XINLINE explicit half(const double& value) { constructor(value); }
  TVM_XINLINE explicit half(const int8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const int32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const long long& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint64_t& value) { constructor(value); }

  TVM_XINLINE operator float() const {                          \
    return float(half2float(half_));                            \
  }                                                             \
  TVM_XINLINE operator float() const volatile {                 \
    return float(half2float(half_));                            \
  }


  TVM_HALF_ASSIGNOP(+=, +)
  TVM_HALF_ASSIGNOP(-=, -)
  TVM_HALF_ASSIGNOP(*=, *)
  TVM_HALF_ASSIGNOP(/=, /)

  TVM_XINLINE half operator+() {
    return *this;
  }

  TVM_XINLINE half operator-() {
    return half(-float(*this));
  }

  TVM_XINLINE half operator=(const half& a) {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) {
    return *this = half(a);
  }

  TVM_XINLINE half operator=(const half& a) volatile {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) volatile {
    return *this = half(a);
  }

 private:
  union Bits {
    float f;
    int32_t si;
    uint32_t ui;
  };

  static int const fp16FractionBits = 10;
  static int const fp32FractionBits = 23;
  static int32_t const fp32FractionMask = ~(~0u << fp32FractionBits);   // == 0x7fffff
  static int32_t const fp32HiddenBit = 1 << fp32FractionBits;   // == 0x800000
  static int const shift = fp32FractionBits - fp16FractionBits;   // == 13
  static int const shiftSign = 16;
  static int32_t const expAdjust = 127 - 15;   // exp32-127 = exp16-15, so exp16 = exp32 - (127-15)

  static int32_t const infN = 0x7F800000;   // flt32 infinity
  static int32_t const maxN = 0x477FFFFF;   // max flt32 that's a flt16 normal after >> by shift
  static int32_t const minN = 0x38800000;   // min flt16 normal as a flt32
  static int32_t const maxZ = 0x33000000;   // max fp32 number that's still rounded to zero in fp16
  static int32_t const signN = 0x80000000;  // flt32 sign bit

  static int32_t const infC = infN >> shift;
  static int32_t const nanN = (infC + 1) << shift;   // minimum flt16 nan as a flt32
  static int32_t const maxC = maxN >> shift;
  static int32_t const minC = minN >> shift;
  static int32_t const signC = signN >> shiftSign;  // flt16 sign bit

  static int32_t const mulN = 0x52000000;  // (1 << 23) / minN
  static int32_t const mulC = 0x33800000;  // minN / (1 << (23 - shift))

  static int32_t const subC = 0x003FF;  // max flt32 subnormal down shifted
  static int32_t const norC = 0x00400;  // min flt32 normal down shifted

  static int32_t const maxD = infC - maxC - 1;
  static int32_t const minD = minC - subC - 1;

  TVM_XINLINE uint16_t float2half(const float& value) const {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  // Same as above routine, except for addition of volatile keyword
  TVM_XINLINE uint16_t float2half(
    const volatile float& value) const volatile {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  TVM_XINLINE float half2float(const uint16_t& value) const {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  TVM_XINLINE float half2float(
    const volatile uint16_t& value) const volatile {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  template<typename T>
  TVM_XINLINE void constructor(const T& value) {
    half_ = float2half(float(value));
  }
};

TVM_HALF_OPERATOR(half, +)
TVM_HALF_OPERATOR(half, -)
TVM_HALF_OPERATOR(half, *)
TVM_HALF_OPERATOR(half, /)
TVM_HALF_OPERATOR(bool, >)
TVM_HALF_OPERATOR(bool, <)
TVM_HALF_OPERATOR(bool, >=)
TVM_HALF_OPERATOR(bool, <=)

TVM_XINLINE half __float2half_rn(const float a) {
  return half(a);
}
#endif


// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

#define CUDA_UNSUPPORTED_HALF_MATH_BINARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x, half y) {   \
  float tmp_x = __half2float(x);                                          \
  float tmp_y = __half2float(y);                                          \
  float result = FP32_MATH_NAME(tmp_x, tmp_y);                            \
  return __float2half(result);                                            \
}

#define CUDA_UNSUPPORTED_HALF_MATH_UNARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x) {          \
  float tmp_x = __half2float(x);                                         \
  float result = FP32_MATH_NAME(tmp_x);                                  \
  return __float2half(result);                                           \
}

// Some fp16 math functions are not supported in hip/hip_fp16.h,
// so we define them here to make sure the generated CUDA code
// is valid.
#if defined(__CUDA_ARCH__)
#if (__CUDA_ARCH__ >= 530)
CUDA_UNSUPPORTED_HALF_MATH_BINARY(hpow, powf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htanh, tanhf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htan, tanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(hatan, atanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(herf, erf)
#else
CUDA_UNSUPPORTED_HALF_MATH_UNARY(hexp, exp)
#endif
#endif

#undef CUDA_UNSUPPORTED_HALF_MATH_BINARY
#undef CUDA_UNSUPPORTED_HALF_MATH_UNARY
__forceinline__ __device__ unsigned int
cast_smem_ptr_to_int(const void* const smem_ptr)
{
  unsigned int smem_int;
  asm volatile ("{ .reg .u64 smem_int; cvta.to.shared.u64 smem_int, %1; cvt.u32.u64 %0, smem_int; }"
    : "=r"(smem_int) : "l"(smem_ptr));
  return smem_int;
}

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 800) 
#define TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST 1
#else
#define TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST 0
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel_opt_m_1024(half* __restrict__ A, half* __restrict__ B, half* __restrict__ C, int m);
extern "C" __global__ void __launch_bounds__(128) main_kernel_opt_m_1024(half* __restrict__ A, half* __restrict__ B, half* __restrict__ C, int m) {
  half C_reindex_pad_shared_warp[32];
  __shared__ half A_reindex_pad_shared[4096];
  __shared__ half B_reindex_shared[4096];
  half A_reindex_pad_shared_warp[16];
  half B_reindex_shared_warp[16];
  half A_reindex_pad_shared_warp_1[16];
  half B_reindex_shared_warp_1[16];
  for (int ax1_0_3_init = 0; ax1_0_3_init < 2; ++ax1_0_3_init) {
    for (int ax2_0_3_init = 0; ax2_0_3_init < 2; ++ax2_0_3_init) {
      for (int i = 0; i < 8; ++i) {
C_reindex_pad_shared_warp[((ax1_0_3_init * 16) + (ax2_0_3_init * 8)) + i] = 0.0;}
;
    }
  }
  #pragma unroll
  for (int ax0_ax1_ax2_fused_2 = 0; ax0_ax1_ax2_fused_2 < 2; ++ax0_ax1_ax2_fused_2) {
    uint4 condval;
    if (((((((((int)blockIdx.y) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 16)) + (ax0_ax1_ax2_fused_2 * 8)) + (((int)threadIdx.x) >> 2)) < m)) {
      condval = *(uint4*)(A + ((((((((int)blockIdx.y) * 65536) + (((int)threadIdx.y) * 32768)) + (((int)threadIdx.z) * 16384)) + (ax0_ax1_ax2_fused_2 * 8192)) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 8)));
    } else {
      condval = make_uint4(__pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)));
    }
    *(uint4*)(A_reindex_pad_shared + (((((((int)threadIdx.y) * 1024) + (((int)threadIdx.z) * 512)) + (ax0_ax1_ax2_fused_2 * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8))) = condval;
  }
  #pragma unroll
  for (int ax0_ax1_ax2_fused_2_1 = 0; ax0_ax1_ax2_fused_2_1 < 2; ++ax0_ax1_ax2_fused_2_1) {
    *(uint4*)(B_reindex_shared + (((((((int)threadIdx.y) * 1024) + (((int)threadIdx.z) * 512)) + (ax0_ax1_ax2_fused_2_1 * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8))) = *(uint4*)(B + ((((((((int)blockIdx.x) * 65536) + (((int)threadIdx.y) * 32768)) + (((int)threadIdx.z) * 16384)) + (ax0_ax1_ax2_fused_2_1 * 8192)) + ((((int)threadIdx.x) >> 2) * 1024)) + ((((int)threadIdx.x) & 3) * 8)));
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax3_0_0 = 0; ax3_0_0 < 31; ++ax3_0_0) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_ax2_fused_2_2 = 0; ax0_ax1_ax2_fused_2_2 < 2; ++ax0_ax1_ax2_fused_2_2) {
      uint4 condval_1;
      if (((((((((int)blockIdx.y) * 64) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.z) * 16)) + (ax0_ax1_ax2_fused_2_2 * 8)) + (((int)threadIdx.x) >> 2)) < m)) {
        condval_1 = *(uint4*)(A + ((((((((((int)blockIdx.y) * 65536) + (((int)threadIdx.y) * 32768)) + (((int)threadIdx.z) * 16384)) + (ax0_ax1_ax2_fused_2_2 * 8192)) + ((((int)threadIdx.x) >> 2) * 1024)) + (ax3_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 32));
      } else {
        condval_1 = make_uint4(__pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)), __pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)));
      }
      *(uint4*)(A_reindex_pad_shared + ((((((((ax3_0_0 + 1) & 1) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.z) * 512)) + (ax0_ax1_ax2_fused_2_2 * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8))) = condval_1;
    }
    #pragma unroll
    for (int ax0_ax1_ax2_fused_2_3 = 0; ax0_ax1_ax2_fused_2_3 < 2; ++ax0_ax1_ax2_fused_2_3) {
      *(uint4*)(B_reindex_shared + ((((((((ax3_0_0 + 1) & 1) * 2048) + (((int)threadIdx.y) * 1024)) + (((int)threadIdx.z) * 512)) + (ax0_ax1_ax2_fused_2_3 * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8))) = *(uint4*)(B + ((((((((((int)blockIdx.x) * 65536) + (((int)threadIdx.y) * 32768)) + (((int)threadIdx.z) * 16384)) + (ax0_ax1_ax2_fused_2_3 * 8192)) + ((((int)threadIdx.x) >> 2) * 1024)) + (ax3_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 32));
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 1;");

    __syncthreads();
    for (int ax3_0_1 = 0; ax3_0_1 < 2; ++ax3_0_1) {
      for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(A_reindex_pad_shared[((((((ax3_0_0 & 1) * 2048) + (((int)threadIdx.y) * 1024)) + (ax0_0 * 512)) + ((((int)threadIdx.x) & 15) * 32)) + ((((ax3_0_1 * 2) + (((int)threadIdx.x) >> 4)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0)));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_reindex_pad_shared[((((((ax3_0_0 & 1) * 2048) + (((int)threadIdx.y) * 1024)) + (ax0_0 * 512)) + ((((int)threadIdx.x) & 15) * 32)) + ((((ax3_0_1 * 2) + (((int)threadIdx.x) >> 4)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_reindex_pad_shared_warp + (ax0_0 * 8)))[0]), "=r"(((unsigned *)(A_reindex_pad_shared_warp + (ax0_0 * 8)))[1]), "=r"(((unsigned *)(A_reindex_pad_shared_warp + (ax0_0 * 8)))[2]), "=r"(((unsigned *)(A_reindex_pad_shared_warp + (ax0_0 * 8)))[3])
      : "r"(addr)
    );
  }
      }
      for (int ax0_0_1 = 0; ax0_0_1 < 2; ++ax0_0_1) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(B_reindex_shared[(((((((ax3_0_0 & 1) * 2048) + (((int)threadIdx.z) * 1024)) + (ax0_0_1 * 512)) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ((((ax3_0_1 * 2) + ((((int)threadIdx.x) & 15) >> 3)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0)));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_reindex_shared[(((((((ax3_0_0 & 1) * 2048) + (((int)threadIdx.z) * 1024)) + (ax0_0_1 * 512)) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ((((ax3_0_1 * 2) + ((((int)threadIdx.x) & 15) >> 3)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))])) + 0))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_reindex_shared_warp + (ax0_0_1 * 8)))[0]), "=r"(((unsigned *)(B_reindex_shared_warp + (ax0_0_1 * 8)))[1]), "=r"(((unsigned *)(B_reindex_shared_warp + (ax0_0_1 * 8)))[2]), "=r"(((unsigned *)(B_reindex_shared_warp + (ax0_0_1 * 8)))[3])
      : "r"(addr)
    );
  }
      }
      for (int ax1_0_3 = 0; ax1_0_3 < 2; ++ax1_0_3) {
        for (int ax2_0_3 = 0; ax2_0_3 < 2; ++ax2_0_3) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_reindex_pad_shared_warp + ((ax1_0_3 * 16) + (ax2_0_3 * 8))))[0]), "=r"(((unsigned *)(C_reindex_pad_shared_warp + ((ax1_0_3 * 16) + (ax2_0_3 * 8))))[1])
      : "r"(((unsigned *)(A_reindex_pad_shared_warp + (ax1_0_3 * 8)))[0]), "r"(((unsigned *)(A_reindex_pad_shared_warp + (ax1_0_3 * 8)))[1]), "r"(((unsigned *)(A_reindex_pad_shared_warp + (ax1_0_3 * 8)))[2]), "r"(((unsigned *)(A_reindex_pad_shared_warp + (ax1_0_3 * 8)))[3]), "r"(((unsigned *)(B_reindex_shared_warp + (ax2_0_3 * 8)))[0]), "r"(((unsigned *)(B_reindex_shared_warp + (ax2_0_3 * 8)))[1]), "r"(((unsigned *)(C_reindex_pad_shared_warp + ((ax1_0_3 * 16) + (ax2_0_3 * 8))))[0]), "r"(((unsigned *)(C_reindex_pad_shared_warp + ((ax1_0_3 * 16) + (ax2_0_3 * 8))))[1]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_reindex_pad_shared_warp + (((ax1_0_3 * 16) + (ax2_0_3 * 8)) + 4)))[0]), "=r"(((unsigned *)(C_reindex_pad_shared_warp + (((ax1_0_3 * 16) + (ax2_0_3 * 8)) + 4)))[1])
      : "r"(((unsigned *)(A_reindex_pad_shared_warp + (ax1_0_3 * 8)))[0]), "r"(((unsigned *)(A_reindex_pad_shared_warp + (ax1_0_3 * 8)))[1]), "r"(((unsigned *)(A_reindex_pad_shared_warp + (ax1_0_3 * 8)))[2]), "r"(((unsigned *)(A_reindex_pad_shared_warp + (ax1_0_3 * 8)))[3]), "r"(((unsigned *)(B_reindex_shared_warp + ((ax2_0_3 * 8) + 4)))[0]), "r"(((unsigned *)(B_reindex_shared_warp + ((ax2_0_3 * 8) + 4)))[1]), "r"(((unsigned *)(C_reindex_pad_shared_warp + (((ax1_0_3 * 16) + (ax2_0_3 * 8)) + 4)))[0]), "r"(((unsigned *)(C_reindex_pad_shared_warp + (((ax1_0_3 * 16) + (ax2_0_3 * 8)) + 4)))[1]));
  }
        }
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int ax3_0_1_1 = 0; ax3_0_1_1 < 2; ++ax3_0_1_1) {
    for (int ax0_0_2 = 0; ax0_0_2 < 2; ++ax0_0_2) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(A_reindex_pad_shared[(((((((int)threadIdx.y) * 1024) + (ax0_0_2 * 512)) + ((((int)threadIdx.x) & 15) * 32)) + ((((ax3_0_1_1 * 2) + (((int)threadIdx.x) >> 4)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8)) + 2048)])) + 0)));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(A_reindex_pad_shared[(((((((int)threadIdx.y) * 1024) + (ax0_0_2 * 512)) + ((((int)threadIdx.x) & 15) * 32)) + ((((ax3_0_1_1 * 2) + (((int)threadIdx.x) >> 4)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8)) + 2048)])) + 0))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax0_0_2 * 8)))[0]), "=r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax0_0_2 * 8)))[1]), "=r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax0_0_2 * 8)))[2]), "=r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax0_0_2 * 8)))[3])
      : "r"(addr)
    );
  }
    }
    for (int ax0_0_3 = 0; ax0_0_3 < 2; ++ax0_0_3) {

  {
    unsigned int addr;
#if TVM_ENBALE_EFFICIENT_SMEM_PTR_CAST
    addr = static_cast<unsigned int>(__cvta_generic_to_shared((void *)((&(B_reindex_shared[((((((((int)threadIdx.z) * 1024) + (ax0_0_3 * 512)) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ((((ax3_0_1_1 * 2) + ((((int)threadIdx.x) & 15) >> 3)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8)) + 2048)])) + 0)));
#else
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(B_reindex_shared[((((((((int)threadIdx.z) * 1024) + (ax0_0_3 * 512)) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ((((ax3_0_1_1 * 2) + ((((int)threadIdx.x) & 15) >> 3)) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8)) + 2048)])) + 0))
    );
#endif
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_reindex_shared_warp_1 + (ax0_0_3 * 8)))[0]), "=r"(((unsigned *)(B_reindex_shared_warp_1 + (ax0_0_3 * 8)))[1]), "=r"(((unsigned *)(B_reindex_shared_warp_1 + (ax0_0_3 * 8)))[2]), "=r"(((unsigned *)(B_reindex_shared_warp_1 + (ax0_0_3 * 8)))[3])
      : "r"(addr)
    );
  }
    }
    for (int ax1_0_3_1 = 0; ax1_0_3_1 < 2; ++ax1_0_3_1) {
      for (int ax2_0_3_1 = 0; ax2_0_3_1 < 2; ++ax2_0_3_1) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_reindex_pad_shared_warp + ((ax1_0_3_1 * 16) + (ax2_0_3_1 * 8))))[0]), "=r"(((unsigned *)(C_reindex_pad_shared_warp + ((ax1_0_3_1 * 16) + (ax2_0_3_1 * 8))))[1])
      : "r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax1_0_3_1 * 8)))[0]), "r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax1_0_3_1 * 8)))[1]), "r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax1_0_3_1 * 8)))[2]), "r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax1_0_3_1 * 8)))[3]), "r"(((unsigned *)(B_reindex_shared_warp_1 + (ax2_0_3_1 * 8)))[0]), "r"(((unsigned *)(B_reindex_shared_warp_1 + (ax2_0_3_1 * 8)))[1]), "r"(((unsigned *)(C_reindex_pad_shared_warp + ((ax1_0_3_1 * 16) + (ax2_0_3_1 * 8))))[0]), "r"(((unsigned *)(C_reindex_pad_shared_warp + ((ax1_0_3_1 * 16) + (ax2_0_3_1 * 8))))[1]));
  }

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n"
      :  "=r"(((unsigned *)(C_reindex_pad_shared_warp + (((ax1_0_3_1 * 16) + (ax2_0_3_1 * 8)) + 4)))[0]), "=r"(((unsigned *)(C_reindex_pad_shared_warp + (((ax1_0_3_1 * 16) + (ax2_0_3_1 * 8)) + 4)))[1])
      : "r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax1_0_3_1 * 8)))[0]), "r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax1_0_3_1 * 8)))[1]), "r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax1_0_3_1 * 8)))[2]), "r"(((unsigned *)(A_reindex_pad_shared_warp_1 + (ax1_0_3_1 * 8)))[3]), "r"(((unsigned *)(B_reindex_shared_warp_1 + ((ax2_0_3_1 * 8) + 4)))[0]), "r"(((unsigned *)(B_reindex_shared_warp_1 + ((ax2_0_3_1 * 8) + 4)))[1]), "r"(((unsigned *)(C_reindex_pad_shared_warp + (((ax1_0_3_1 * 16) + (ax2_0_3_1 * 8)) + 4)))[0]), "r"(((unsigned *)(C_reindex_pad_shared_warp + (((ax1_0_3_1 * 16) + (ax2_0_3_1 * 8)) + 4)))[1]));
  }
      }
    }
  }
  for (int ax0_1 = 0; ax0_1 < 2; ++ax0_1) {
    for (int ax1 = 0; ax1 < 2; ++ax1) {
      __syncthreads();
      for (int local_id = 0; local_id < 8; local_id+=2) {
*((uint *)&(&(A_reindex_pad_shared[((((int)threadIdx.y) * 1536) + (((int)threadIdx.z) * 512))]))[((((((local_id % 4) / 2) * 8) + (threadIdx.x / 4)) * 16) + ((((local_id / 4) * 8) + ((threadIdx.x % 4) * 2)) + (local_id % 2)))]) = *((uint *)&C_reindex_pad_shared_warp[((ax0_1 * 16) + (ax1 * 8)) + local_id]);
}
;
      __syncthreads();
      #pragma unroll
      for (int ax0_ax1_ax2_ax3_ax4_fused_0 = 0; ax0_ax1_ax2_ax3_ax4_fused_0 < 1; ++ax0_ax1_ax2_ax3_ax4_fused_0) {
        if (((((((int)blockIdx.y) * 64) + (((int)threadIdx.y) * 32)) + (ax0_1 * 16)) + (((int)threadIdx.x) >> 1)) < m) {
          *(uint4*)(C + ((((((((((int)blockIdx.y) * 65536) + (((int)threadIdx.y) * 32768)) + (ax0_1 * 16384)) + ((((int)threadIdx.x) >> 1) * 1024)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.z) * 32)) + (ax1 * 16)) + ((((int)threadIdx.x) & 1) * 8))) = *(uint4*)(A_reindex_pad_shared + (((((int)threadIdx.y) * 1536) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.x) * 8)));
        }
      }
    }
  }
}

